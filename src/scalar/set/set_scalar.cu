#include "hip/hip_runtime.h"
#include <scalar.h>

__device__ double op(double d1,double d2,double *params) {
   return d2;
}

extern "C"
__global__ void set_scalar_double(int n, int idx,double dx,double *dy,int incy,double *params,double *result,int blockSize) {
       transform<double>(n,idx,dx,dy,incy,params,result,blockSize);
 }


__device__ float op(float d1,float d2,float *params) {
   return d2;
}

extern "C"
__global__ void set_scalar_float(int n, int idx,float dx,float *dy,int incy,float *params,float *result,int blockSize) {
       transform<float>(n,idx,dx,dy,incy,params,result,blockSize);
 }
