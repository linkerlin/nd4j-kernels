#include "hip/hip_runtime.h"
#include <scalar.h>
//scalar and current element
__device__ double op(double d1,double d2,double *params) {
	if(d2 == d1) {
		return 1;
	}
	return 0;

}

extern "C"
__global__ void equals_scalar_double(int n, int idx,double dx,double *dy,int incx,double *params,double *result,int blockSize) {
	transform<double>(n,idx,dx,dy,incx,params,result,blockSize);
}


__device__ float op(float d1,float d2,float *params) {
	if(d2 == d1) {
		return 1;
	}
	return 0;

}

extern "C"
__global__ void equals_scalar_float(int n, int idx,float dx,float *dy,int incx,float *params,float *result,int blockSize) {
	transform<float>(n,idx,dx,dy,incx,params,result,blockSize);
}

