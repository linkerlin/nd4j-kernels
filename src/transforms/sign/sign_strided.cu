#include "hip/hip_runtime.h"
#include <transform.h>


template<> __device__ double op<double>(double d1,double *params) {
	return (d1 > 0) - (d1 < 0);
}

extern "C"
__global__ void sign_strided_double(int n,int idx,double *dy,int incy,double *params,double *result,int blockSize) {
	transform<double>(n,idx,dy,incy,params,result,blockSize);

}


template<> __device__ float op<float>(float d1,float *params) {
	return (d1 > 0) - (d1 < 0);
}

extern "C"
__global__ void sign_strided_float(int n,int idx,float *dy,int incy,float *params,float *result,int blockSize) {
	transform<float>(n,idx,dy,incy,params,result,blockSize);

}
