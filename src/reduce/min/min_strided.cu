#include "hip/hip_runtime.h"
#include <reduce.h>
template<> __device__ double update<double>(double old,double opOutput,double *extraParams) {
	return min(old,opOutput);
}

template<> __device__ double merge<double>(double old,double opOutput,double *extraParams) {
	return min(old,opOutput);
}

/**
 An op on the device
 @param d1 the first operator
 @param d2 the second operator
 */
template<> __device__ double op<double>(double d1,double d2,double *extraParams) {
	return d1;
}

template<> __device__ double op<double>(double d1,double *extraParams) {
	return d1;
}



template<> __device__ double postProcess<double>(double reduction,int n,int xOffset,double *dx,int incx,double *extraParams,double *result) {
	return reduction;
}

__global__ void min_strided_double(	int n
		,double *dx
		,int *xVectorInfo
		,double *extraParams
		,double *result,
		int *resultVectorInfo
		,int *gpuInformation,
		int *problemDefinition) {
	transform<double>(n,dx,xVectorInfo,extraParams,result,resultVectorInfo,gpuInformation,problemDefinition);
}


template<> __device__ float update<float>(float old,float opOutput,float *extraParams) {
	return fminf(old,opOutput);
}

template<> __device__ float merge<float>(float old,float opOutput,float *extraParams) {
	return fminf(old,opOutput);
}

/**
 An op on the device
 @param d1 the first operator
 @param d2 the second operator
 */
template<> __device__ float op<float>(float d1,float d2,float *extraParams) {
	return d1;
}

template<> __device__ float op<float>(float d1,float *extraParams) {
	return d1;
}



template<> __device__ float postProcess<float>(float reduction,int n,int xOffset,float *dx,int incx,float *extraParams,float *result) {
	return reduction;
}

__global__ void min_strided_float(	int n
		,float *dx
		,int *xVectorInfo
		,float *extraParams
		,float *result,
		int *resultVectorInfo
		,int *gpuInformation,
		int *problemDefinition) {
	transform<float>(n,dx,xVectorInfo,extraParams,result,resultVectorInfo,gpuInformation,problemDefinition);
}

