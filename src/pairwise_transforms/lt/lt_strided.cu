#include "hip/hip_runtime.h"
#include <pairwise_transform.h>

__device__ double op(double d1,double d2,double *params) {
	if(d1 < d2) return 1;
	else return 0;
}
__device__ double op(double d1,double *params) {
	return d1;
}

extern "C"
__global__ void lt_strided_double(int n,int xOffset,int yOffset, double *dx, double *dy,int incx,int incy,double *params,double *result,int incz,int blockSize) {
	transform<double>(n,xOffset,yOffset,dx,dy,incx,incy,params,result,incz,blockSize);
}


__device__ float op(float d1,float d2,float *params) {
	if(d1 < d2) return 1;
	else return 0;
}
__device__ float op(float d1,float *params) {
	return d1;
}

extern "C"
__global__ void lt_strided_float(int n,int xOffset,int yOffset, float *dx, float *dy,int incx,int incy,float *params,float *result,int incz,int blockSize) {
	transform<float>(n,xOffset,yOffset,dx,dy,incx,incy,params,result,incz,blockSize);
}
