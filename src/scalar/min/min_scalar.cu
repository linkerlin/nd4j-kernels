#include "hip/hip_runtime.h"
#include <scalar.h>

template<> __device__ double op<double>(double d1,double d2,double *params) {
	if(d1 < d2)
		return 1;
	return 0;
}


__global__ void min_scalar_double(int n, int idx,double dx,double *dy,int incy,double *params,double *result,int blockSize) {
	transform<double>(n,idx,dx,dy,incy,params,result,blockSize);
}


template<> __device__ float op<float>(float d1,float d2,float *params) {
	if(d1 < d2)
		return 1;
	return 0;
}


__global__ void min_scalar_float(int n, int idx,float dx,float *dy,int incy,float *params,float *result,int blockSize) {
	transform<float>(n,idx,dx,dy,incy,params,result,blockSize);
}
