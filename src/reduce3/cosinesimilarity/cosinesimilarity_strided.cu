#include "hip/hip_runtime.h"
#include <reduce3.h>


template<> __device__ double update<double>(double old,double opOutput,double *extraParams) {
	return old + opOutput;
}


/**
 An op on the device
 @param d1 the first operator
 @param d2 the second operator
 */
template<> __device__ double op<double>(double d1,double d2,double *extraParams) {
	return d1 * d2;
}


//post process result (for things like means etc)
template<> __device__ double postProcess<double>(double reduction,int n,int xOffset,double *dx,int incx,double *extraParams,double *result) {
	return reduction / extraParams[1] / extraParams[2];
}


__global__ void cosinesimilarity_strided_double(int n, int xOffset,int yOffset,double *dx,double *dy,int incx,int incy,double *extraParams,double *result,int i,int blockSize) {
	transform_pair<double>(n,xOffset,yOffset,dx,dy,incx,incy,extraParams,result,i,blockSize);

}



template<> __device__ float update<float>(float old,float opOutput,float *extraParams) {
	return old + opOutput;
}


/**
 An op on the device
 @param d1 the first operator
 @param d2 the second operator
 */
template<> __device__ float op<float>(float d1,float d2,float *extraParams) {
	return d1 * d2;
}


//post process result (for things like means etc)
template<> __device__ float postProcess<float>(float reduction,int n,int xOffset,float *dx,int incx,float *extraParams,float *result) {
	return reduction / extraParams[1] / extraParams[2];
}


__global__ void cosinesimilarity_strided_float(int n, int xOffset,int yOffset,float *dx,float *dy,int incx,int incy,float *extraParams,float *result,int i,int blockSize) {
	transform_pair<float>(n,xOffset,yOffset,dx,dy,incx,incy,extraParams,result,i,blockSize);

}



