#include "hip/hip_runtime.h"
#include <scalar.h>

template<> __device__ double op<double>(double d1,double d2,double *params) {
   return d2;
}


__global__ void set_scalar_double(int n, int idx,double dx,double *dy,int incy,double *params,double *result,int blockSize) {
       transform<double>(n,idx,dx,dy,incy,params,result,blockSize);
 }


template<> __device__ float op<float>(float d1,float d2,float *params) {
   return d2;
}


__global__ void set_scalar_float(int n, int idx,float dx,float *dy,int incy,float *params,float *result,int blockSize) {
       transform<float>(n,idx,dx,dy,incy,params,result,blockSize);
 }
