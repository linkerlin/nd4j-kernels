#include "hip/hip_runtime.h"
#include <transform.h>


template<> __device__ double op<double>(double d1,double *params) {
        return sin(d1);
}


__global__ void sin_strided_double(int n,int idx,double *dy,int incy,double *params,double *result,int blockSize) {
       transform<double>(n,idx,dy,incy,params,result,blockSize);

 }


template<> __device__ float op<float>(float d1,float *params) {
        return sinf(d1);
}


__global__ void sin_strided_float(int n,int idx,float *dy,int incy,float *params,float *result,int blockSize) {
       transform<float>(n,idx,dy,incy,params,result,blockSize);

 }
