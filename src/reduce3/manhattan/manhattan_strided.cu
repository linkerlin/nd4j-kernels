#include "hip/hip_runtime.h"
#include <reduce3.h>


__device__ double update(double old,double opOutput,double *extraParams) {
	return pow(old,2) + opOutput;
}


/**
 An op on the device
 @param d1 the first operator
 @param d2 the second operator
 */
__device__ double op(double d1,double d2,double *extraParams) {
	return d1 - d2;
}


//post process result (for things like means etc)
__device__ double postProcess(double reduction,int n,int xOffset,double *dx,int incx,double *extraParams,double *result) {
	return reduction / extraParams[0] / extraParams[1];
}

extern "C"
__global__ void manhattan_strided_double(int n, int xOffset,int yOffset,double *dx,double *dy,int incx,int incy,double *extraParams,double *result,int i,int blockSize) {
	transform_pair<double>(n,xOffset,yOffset,dx,dy,incx,incy,extraParams,result,i,blockSize);

}



__device__ float update(float old,float opOutput,float *extraParams) {
	return pow(old,2) + opOutput;
}


/**
 An op on the device
 @param d1 the first operator
 @param d2 the second operator
 */
__device__ float op(float d1,float d2,float *extraParams) {
	return d1 - d2;
}


//post process result (for things like means etc)
__device__ float postProcess(float reduction,int n,int xOffset,float *dx,int incx,float *extraParams,float *result) {
	return reduction / extraParams[0] / extraParams[1];
}

extern "C"
__global__ void manhattan_strided_float(int n, int xOffset,int yOffset,float *dx,float *dy,int incx,int incy,float *extraParams,float *result,int i,int blockSize) {
	transform_pair<float>(n,xOffset,yOffset,dx,dy,incx,incy,extraParams,result,i,blockSize);

}
