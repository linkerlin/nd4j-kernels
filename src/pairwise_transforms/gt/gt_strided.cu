#include "hip/hip_runtime.h"
#include <pairwise_transform.h>

template <>  __device__ double op<double>(double d1,double d2,double *params) {
	if(d1 > d2) return 1;
	else return 0;
}

template <> __device__ double op<double>(double d1,double *params) {
	return d1;
}


__global__ void gt_strided_double(int n,int xOffset,int yOffset, double *dx, double *dy,int incx,int incy,double *params,double *result,int incz,int blockSize) {
	transform<double>(n,xOffset,yOffset,dx,dy,incx,incy,params,result,incz,blockSize);
}

template <>  __device__ float op<float>(float d1,float d2,float *params) {
	if(d1 > d2) return 1;
	else return 0;
}

template<> __device__ float op<float>(float d1,float *params) {
	return d1;
}


__global__ void gt_strided_float(int n,int xOffset,int yOffset, float *dx, float *dy,int incx,int incy,float *params,float *result,int incz,int blockSize) {
	transform<float>(n,xOffset,yOffset,dx,dy,incx,incy,params,result,incz,blockSize);
}
