#include "hip/hip_runtime.h"
#include <reduce3.h>


__device__ double update(double old,double opOutput,double *extraParams) {
	return old + opOutput;
}


/**
 An op on the device
 @param d1 the first operator
 @param d2 the second operator
 */
__device__ double op(double d1,double d2,double *extraParams) {
	return d1 * d2;
}


//post process result (for things like means etc)
__device__ double postProcess(double reduction,int n,int xOffset,double *dx,int incx,double *extraParams,double *result) {
	return reduction / extraParams[1] / extraParams[2];
}

extern "C"
__global__ void cosinesimilarity_strided_double(int n, int xOffset,int yOffset,double *dx,double *dy,int incx,int incy,double *extraParams,double *result,int i,int blockSize) {
	transform_pair<double>(n,xOffset,yOffset,dx,dy,incx,incy,extraParams,result,i,blockSize);

}



__device__ float update(float old,float opOutput,float *extraParams) {
	return old + opOutput;
}


/**
 An op on the device
 @param d1 the first operator
 @param d2 the second operator
 */
__device__ float op(float d1,float d2,float *extraParams) {
	return d1 * d2;
}


//post process result (for things like means etc)
__device__ float postProcess(float reduction,int n,int xOffset,float *dx,int incx,float *extraParams,float *result) {
	return reduction / extraParams[1] / extraParams[2];
}

extern "C"
__global__ void cosinesimilarity_strided_float(int n, int xOffset,int yOffset,float *dx,float *dy,int incx,int incy,float *extraParams,float *result,int i,int blockSize) {
	transform_pair<float>(n,xOffset,yOffset,dx,dy,incx,incy,extraParams,result,i,blockSize);

}



