#include "hip/hip_runtime.h"
#include <transform.h>


__device__ double op(double d1,double *params) {
        return pow(d1,params[0]);
}

extern "C"
__global__ void pow_strided_double(int n,int idx,double *dy,int incy,double *params,double *result,int blockSize) {
       transform<double>(n,idx,dy,incy,params,result,blockSize);

 }

__device__ float op(float d1,float *params) {
        return powf(d1,params[0]);
}

extern "C"
__global__ void pow_strided_float(int n,int idx,float *dy,int incy,float *params,float *result,int blockSize) {
       transform<float>(n,idx,dy,incy,params,result,blockSize);

 }
